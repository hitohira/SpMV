#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <stdio.h>
#include "matrix.h"


// see toolkit document v10.0
template<typename T>
void CSR<T>::CuSparseMul(Vec<T>& x, Vec<T>& y){
}
template<> void CSR<float>::CuSparseMul(Vec<float>& x,Vec<float>& y){
	hipsparseHandle_t handle;
	hipsparseMatDescr_t descr;

	float alpha = 1;
	float beta = 0;

	hipsparseCreate(&handle);
	hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

	hipsparseScsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,m,n,rowptr[m],
	                &alpha,descr,d_val,d_rowptr,d_colind,x.d_val,&beta,y.d_val);

	hipsparseDestroyMatDescr(descr);
	hipsparseDestroy(handle);
}
template<> void CSR<double>::CuSparseMul(Vec<double>& x,Vec<double>& y){
	hipsparseHandle_t handle;
	hipsparseMatDescr_t descr;

	double alpha = 1;
	double beta = 0;

	hipsparseCreate(&handle);
	hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

	hipsparseDcsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,m,n,rowptr[m],
	                &alpha,descr,d_val,d_rowptr,d_colind,x.d_val,&beta,y.d_val);

	hipsparseDestroyMatDescr(descr);
	hipsparseDestroy(handle);
}
