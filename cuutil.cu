#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "matrix.h"

template<typename T>
void FreeDeviceMemory(T* d_ptr){
	hipFree(d_ptr);
}
template void FreeDeviceMemory(float* d_ptr);
template void FreeDeviceMemory(double* d_ptr);
template void FreeDeviceMemory(int* d_ptr);

template<typename T>
void CSR<T>::CopyMatToDevice(){
	  int nnz = rowptr[m];
		if(d_val) hipFree(d_val);
		if(d_rowptr) hipFree(d_rowptr);
		if(d_colind) hipFree(d_colind);
		 hipMalloc((void**)&d_val,nnz*sizeof(T));
		 hipMalloc((void**)&d_colind,nnz*sizeof(int));
		 hipMalloc((void**)&d_rowptr,(m+1)*sizeof(int));
		 hipMemcpy(d_val,val,nnz*sizeof(T),hipMemcpyHostToDevice);
		 hipMemcpy(d_colind,colind,nnz*sizeof(int),hipMemcpyHostToDevice);
		 hipMemcpy(d_rowptr,rowptr,(m+1)*sizeof(int),hipMemcpyHostToDevice);
}
template void CSR<float>::CopyMatToDevice();
template void CSR<double>::CopyMatToDevice();

template<typename T>
void Vec<T>::AllocVectorToDevice(){
	if(d_val) hipFree(d_val);
	hipMalloc((void**)&d_val,m*sizeof(T));
}
template void Vec<float>::AllocVectorToDevice();
template void Vec<double>::AllocVectorToDevice();

template<typename T>
void Vec<T>::SetVectorValueToDevice(){
	hipMemcpy(d_val,val,m*sizeof(T),hipMemcpyHostToDevice);
}
template void Vec<float>::SetVectorValueToDevice();
template void Vec<double>::SetVectorValueToDevice();

template<typename T>
void Vec<T>::GetVectorValueFromDevice(){
	hipMemcpy(val,d_val,m*sizeof(T),hipMemcpyDeviceToHost);
}
template void Vec<float>::GetVectorValueFromDevice();
template void Vec<double>::GetVectorValueFromDevice();


template<typename T>
void ELL<T>::CopyMatToDevice(){
	if(d_val) hipFree(d_val);
	if(d_colind) hipFree(d_colind);
	hipMalloc((void**)&d_val,m*k*sizeof(T));
	hipMalloc((void**)&d_colind,m*k*sizeof(int));

	hipMemcpy(d_val,val,m*k*sizeof(T),hipMemcpyHostToDevice);
	hipMemcpy(d_colind,colind,m*k*sizeof(T),hipMemcpyHostToDevice);
}
template void ELL<float>::CopyMatToDevice();
template void ELL<double>::CopyMatToDevice();

