#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "matrix.h"

template<typename T>
void FreeDeviceMemory(T* d_ptr){
	hipFree(d_ptr);
}
template void FreeDeviceMemory(float* d_ptr);
template void FreeDeviceMemory(double* d_ptr);
template void FreeDeviceMemory(int* d_ptr);


template<typename T>
int Vec<T>::AllocVectorToDevice(){
	if(d_val) hipFree(d_val);
	if(hipMalloc((void**)&d_val,m*sizeof(T)) == hipSuccess){
		return 0;
	}
	else{
		return -1;
	}
}
template int Vec<float>::AllocVectorToDevice();
template int Vec<double>::AllocVectorToDevice();

template<typename T>
int Vec<T>::SetVectorValueToDevice(){
	if(hipMemcpy(d_val,val,m*sizeof(T),hipMemcpyHostToDevice) == hipSuccess){
		return 0;
	}
	return -1;
}
template int Vec<float>::SetVectorValueToDevice();
template int Vec<double>::SetVectorValueToDevice();

template<typename T>
int Vec<T>::GetVectorValueFromDevice(){
	if(hipMemcpy(val,d_val,m*sizeof(T),hipMemcpyDeviceToHost) == hipSuccess){
		return 0;
	}
	return -1;
}
template int Vec<float>::GetVectorValueFromDevice();
template int Vec<double>::GetVectorValueFromDevice();



template<typename T>
int CSR<T>::CopyMatToDevice(){
		hipError_t err;
	  int nnz = rowptr[m];
		if(d_val) hipFree(d_val);
		if(d_rowptr) hipFree(d_rowptr);
		if(d_colind) hipFree(d_colind);
		 err = hipMalloc((void**)&d_val,nnz*sizeof(T));
		 if(err != hipSuccess){ fprintf(stderr,"fail to malloc on GPU\n");return -1;}
		 err = hipMalloc((void**)&d_colind,nnz*sizeof(int));
		 if(err != hipSuccess){ fprintf(stderr,"fail to malloc on GPU\n");return -1;}
		 err = hipMalloc((void**)&d_rowptr,(m+1)*sizeof(int));
		 if(err != hipSuccess){ fprintf(stderr,"fail to malloc on GPU\n");return -1;}
		 err = hipMemcpy(d_val,val,nnz*sizeof(T),hipMemcpyHostToDevice);
		 if(err != hipSuccess){ fprintf(stderr,"fail to memcpy to GPU\n");return -1;}
		 err = hipMemcpy(d_colind,colind,nnz*sizeof(int),hipMemcpyHostToDevice);
		 if(err != hipSuccess){ fprintf(stderr,"fail to memcpy to GPU\n");return -1;}
		 err = hipMemcpy(d_rowptr,rowptr,(m+1)*sizeof(int),hipMemcpyHostToDevice);
		 if(err != hipSuccess){ fprintf(stderr,"fail to memcpy to GPU\n");return -1;}
		 return 0;
}
template int CSR<float>::CopyMatToDevice();
template int CSR<double>::CopyMatToDevice();


template<typename T>
int ELL<T>::CopyMatToDevice(){
	hipError_t err;
	if(d_val) hipFree(d_val);
	if(d_colind) hipFree(d_colind);
	err = hipMalloc((void**)&d_val,m*k*sizeof(T));
	if(err != hipSuccess){ fprintf(stderr,"fail to malloc on GPU\n");return -1;}
	err = hipMalloc((void**)&d_colind,m*k*sizeof(int));
	if(err != hipSuccess){ fprintf(stderr,"fail to malloc on GPU\n");return -1;}

	err = hipMemcpy(d_val,val,m*k*sizeof(T),hipMemcpyHostToDevice);
	if(err != hipSuccess){ fprintf(stderr,"fail to memcpy to GPU\n");return -1;}
	err = hipMemcpy(d_colind,colind,m*k*sizeof(T),hipMemcpyHostToDevice);
	if(err != hipSuccess){ fprintf(stderr,"fail to memcpy to GPU\n");return -1;}
	return 0;
}
template int ELL<float>::CopyMatToDevice();
template int ELL<double>::CopyMatToDevice();

template<typename T>
int COO<T>::CopyMatToDevice(){
	hipError_t err;
	if(d_val) hipFree(d_val);
	if(d_colind) hipFree(d_colind);
	if(d_rowind) hipFree(d_rowind);
	err = hipMalloc((void**)&d_val,nnz*sizeof(T));
	if(err != hipSuccess) {fprintf(stderr,"fail at malloc on GPU\n");return -1;}
	err = hipMalloc((void**)&d_colind,nnz*sizeof(T));
	if(err != hipSuccess) {fprintf(stderr,"fail at malloc on GPU\n");return -1;}
	err = hipMalloc((void**)&d_rowind,nnz*sizeof(T));
	if(err != hipSuccess) {fprintf(stderr,"fail at malloc on GPU\n");return -1;}

	err = hipMemcpy(d_val,val,nnz*sizeof(T),hipMemcpyHostToDevice);
	if(err != hipSuccess) { fprintf(stderr,"fail at memcpy to GPU\n");return -1;}
	err = hipMemcpy(d_colind,colind,nnz*sizeof(T),hipMemcpyHostToDevice);
	if(err != hipSuccess) { fprintf(stderr,"fail at memcpy to GPU\n");return -1;}
	err = hipMemcpy(d_rowind,rowind,nnz*sizeof(T),hipMemcpyHostToDevice);
	if(err != hipSuccess) { fprintf(stderr,"fail at memcpy to GPU\n");return -1;}
	return 0;
}
template int COO<float>::CopyMatToDevice();
template int COO<double>::CopyMatToDevice();
