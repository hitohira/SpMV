#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "matrix.h"

__global__ void EllKernelS(const int nrow,const int width,
                                const int* col,const float* val, const float* b,float* c){
	int r = blockDim.x * blockIdx.x + threadIdx.x;
	if(r < nrow){
		float dot = 0;

		for(int i = 0; i < width; i++){
			int idx = col[nrow*i+r];
			float v = val[nrow*i+r];
			dot += v * b[idx];
		}
		c[r] = dot;
	}
}


template<typename X>
void ELL<X>::MulOnGPU(Vec<X>& x, Vec<X>& y){
}
template<> void ELL<float>::MulOnGPU(Vec<float>& x, Vec<float>& y){
	int T = 1024;
	int B = m / T + 1;
	EllKernelS<<<B,T>>>(m,k,d_colind,d_val,x.d_val,y.d_val);
}
template<> void ELL<double>::MulOnGPU(Vec<double>& x, Vec<double>& y){

}


__global__ void EllKernelSWithTex(const int nrow,const int width,
                                const int* col,const float* val, hipTextureObject_t b,float* c){
	int r = blockDim.x * blockIdx.x + threadIdx.x;
	if(r < nrow){
		float dot = 0;

		for(int i = 0; i < width; i++){
			int idx = col[nrow*i+r];
			float v = val[nrow*i+r];
			if(v != 0){
				float bval = tex1Dfetch<float>(b,idx);
				dot += v * bval;
			}
		}
		c[r] = dot;
	}
}


template<typename X>
void ELL<X>::MulOnGPUWithTex(Vec<X>& x, Vec<X>& y){
}
template<> void ELL<float>::MulOnGPUWithTex(Vec<float>& x, Vec<float>& y){
	int T = 1024;
	int B = m / T + 1;
	EllKernelSWithTex<<<B,T>>>(m,k,d_colind,d_val,*((hipTextureObject_t*)(x.texVec.texObj)),y.d_val);
}
template<> void ELL<double>::MulOnGPUWithTex(Vec<double>& x, Vec<double>& y){

}

