#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "tex.h"

template<typename T>
void TexVec<T>::Free(){
	if(cuArray){
		hipFreeArray((hipArray*)cuArray);
	}
	if(texObj){
		hipDestroyTextureObject(*(hipTextureObject_t*)texObj);
		free(texObj);
	}
}
template void TexVec<float>::Free();
template void TexVec<double>::Free();


template<typename T>
void TexVec<T>::SetTexVec(int m,T* data){
}
template<> void TexVec<float>::SetTexVec(int m,float* data){
	hipError_t err;
	Free();
	texObj = NULL;
	cuArray = NULL;

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindFloat);
	int size = m * sizeof(float);
/*
	int width = m;
	int height = 1;
	err = hipMallocArray(((hipArray**)(&cuArray)),&channelDesc,width,height);
	if(err != hipSuccess){ fprintf(stderr,"hipMallocArray failed; size=%d\n",m);return; }
	err = hipMemcpyToArray((hipArray*)cuArray,0,0,data,size,hipMemcpyHostToDevice);
	if(err != hipSuccess){ fprintf(stderr,"hipMemcpyToArray failed\n");return; }
*/

	struct hipResourceDesc resDesc;
	memset(&resDesc,0,sizeof(resDesc));
//	resDesc.resType = hipResourceTypeArray;
//	resDesc.res.array.array = (hipArray*)cuArray;
	resDesc.resType = hipResourceTypeLinear;
	resDesc.res.linear.devPtr = (void*)data;
	resDesc.res.linear.desc = channelDesc;
	resDesc.res.linear.sizeInBytes = size;

	struct hipTextureDesc texDesc;
	memset(&texDesc,0,sizeof(texDesc));
	texDesc.addressMode[0]   = hipAddressModeWrap;
	texDesc.addressMode[1]   = hipAddressModeWrap;
	texDesc.filterMode       = hipFilterModePoint;
	texDesc.readMode         = hipReadModeElementType;
	texDesc.normalizedCoords = 0;

	texObj = malloc(sizeof(hipTextureObject_t));
	err = hipCreateTextureObject((hipTextureObject_t*)texObj,&resDesc,&texDesc,NULL);
	if(err != hipSuccess){ fprintf(stderr,"hipCreateTextureObject failed\n");return; }

}
template<> void TexVec<double>::SetTexVec(int m,double* data){
	Free();
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32,32,0,0,hipChannelFormatKindSigned);

}
