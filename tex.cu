#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "tex.h"

template<typename T>
void TexVec<T>::Free(){
	if(cuArray){
		hipFreeArray((hipArray*)cuArray);
	}
	if(texObj){
		hipDestroyTextureObject(*(hipTextureObject_t*)texObj);
		free(texObj);
	}
}
template void TexVec<float>::Free();
template void TexVec<double>::Free();


template<typename T>
void TexVec<T>::SetTexVec(int m,T* data){
}
template<> void TexVec<float>::SetTexVec(int m,float* data){
	Free();
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindFloat);
	int size = m * sizeof(float);
	int width = m;
	int height = 1;
	hipMallocArray(((hipArray**)(&cuArray)),&channelDesc,width,height);
	hipMemcpyToArray((hipArray*)cuArray,0,0,data,size,hipMemcpyHostToDevice);

	struct hipResourceDesc resDesc;
	memset(&resDesc,0,sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = (hipArray*)cuArray;

	struct hipTextureDesc texDesc;
	memset(&texDesc,0,sizeof(texDesc));
	texDesc.addressMode[0]   = hipAddressModeWrap;
	texDesc.addressMode[1]   = hipAddressModeWrap;
	texDesc.filterMode       = hipFilterModePoint;
	texDesc.readMode         = hipReadModeElementType;
	texDesc.normalizedCoords = 0;

	texObj = malloc(sizeof(hipTextureObject_t));
	hipCreateTextureObject((hipTextureObject_t*)texObj,&resDesc,&texDesc,NULL);

}
template<> void TexVec<double>::SetTexVec(int m,double* data){
	Free();
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32,32,0,0,hipChannelFormatKindSigned);

}
